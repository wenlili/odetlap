#include "hip/hip_runtime.h"
#include "approximator.h"

// Append a nonzero element to A
static void append(cusp::coo_matrix<int,float,HOST> &A, const int index, const int row, const int col, const float val)
{
    A.row_indices[index] = row;
    A.column_indices[index] = col;
    A.values[index] = val;
}

// Approximate data from known points
bool approximate(cusp::array1d<float,HOST> &h_x, const int nslcs, const int nrows, const int ncols, const cusp::array1d<float4,HOST> &known, const float R)
{
    const int size = nslcs*nrows*ncols; // number of points
    const int m = size + known.size();  // number of equations
    const int n = size;                 // number of variables
    const int nnz = 7*size - 2*(nslcs*nrows + nslcs*ncols + nrows*ncols) + known.size(); // number of nonzero elements

    // Build A
    cusp::coo_matrix<int,float,HOST> h_A(m, n, nnz);
    int index = 0;
    for (int s = 0; s < nslcs; s++)
        for (int r = 0; r < nrows; r++)
            for (int c = 0; c < ncols; c++) {
                int i = s*nrows*ncols + r*ncols + c;
                float sum = 0;
                if (s > 0)       { append(h_A, index++, i, i-nrows*ncols, -R); sum += R; }
                if (s < nslcs-1) { append(h_A, index++, i, i+nrows*ncols, -R); sum += R; }
                if (r > 0)       { append(h_A, index++, i, i-ncols, -R);       sum += R; }
                if (r < nrows-1) { append(h_A, index++, i, i+ncols, -R);       sum += R; }
                if (c > 0)       { append(h_A, index++, i, i-1, -R);           sum += R; }
                if (c < ncols-1) { append(h_A, index++, i, i+1, -R);           sum += R; }
                append(h_A, index++, i, i, sum);
            }
    for (int i = 0; i < known.size(); i++)
        append(h_A, index++, n+i, int(known[i].x)*nrows*ncols+int(known[i].y)*ncols+int(known[i].z), 1);
    cusp::coo_matrix<int,float,DEVICE> A(h_A);

    // Build b
    cusp::array1d<float,HOST> h_b(m, 0);
    for (int i = 0; i < known.size(); i++)
        h_b[n+i] = known[i].w;
    cusp::array1d<float,DEVICE> b(h_b);

    // Solve Ax = b
    cusp::array1d<float,DEVICE> x(n, 0);
    bool result = solve(A, x, b);
    h_x = x;
    return result;
}
