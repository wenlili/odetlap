#include "hip/hip_runtime.h"
#include <fstream>
#include <sstream>
#include <algorithm>
#include <string>
#include "approximator.h"

#define SW 12
#define OW 6
#define RW 4
#define IT 1
#define R 0.01

// Convert coordinates to an index
inline int idx(int s, int r, int c, int nr, int nc)
{
    return s*nr*nc + r*nc + c;
}

// Process a segment
int process_segment(const float target, const int ss, const int sr, const int sc, cusp::array1d<bool,HOST> &pending, const int nslcs, const int nrows, const int ncols,
                    const cusp::array1d<float,HOST> &data, const cusp::array1d<short,HOST> &nzmask, cusp::array1d<short,HOST> &mask, cusp::array1d<float,HOST> &approximation)
{
    int count = 0;
    const int sns = nslcs/SW;
    const int snr = nrows/SW;
    const int snc = ncols/SW;
    if (!pending[idx(ss,sr,sc,snr,snc)])
        return count;
    
    const int ns0 = max(ss*SW-OW, 0);
    const int ns1 = min((ss+1)*SW+OW, nslcs);
    const int nr0 = max(sr*SW-OW, 0);
    const int nr1 = min((sr+1)*SW+OW, nrows);
    const int nc0 = max(sc*SW-OW, 0);
    const int nc1 = min((sc+1)*SW+OW, ncols);
    const int nns = ns1 - ns0;
    const int nnr = nr1 - nr0;
    const int nnc = nc1 - nc0;
    const int ds = ss*SW - ns0;
    const int dr = sr*SW - nr0;
    const int dc = sc*SW - nc0;
    
    cusp::array1d<float4,HOST> known;
    for (int i = ns0; i < ns1; i++)
        for (int j = nr0; j < nr1; j++)
            for (int k = nc0; k < nc1; k++)
                if (mask[idx(i,j,k,nrows,ncols)])
                    known.push_back(make_float4(i-ns0, j-nr0, k-nc0, data[idx(i,j,k,nrows,ncols)]));
    cusp::array1d<float,HOST> x(nns*nnr*nnc);
    
    for (int it = 0; it < IT; it++) {
        if (approximate(x, nns, nnr, nnc, known, R))
            ; //std::cout << 'c';
        else
            std::cout << '-';
        float maxe = 0;
        int sw = 0;
        int rw = 0;
        int cw = 0;
        for (int i = 0; i < SW; i++)
            for (int j = 0; j < SW; j++)
                for (int k = 0; k < SW; k++)
                    if (nzmask[idx(ss*SW+i,sr*SW+j,sc*SW+k,nrows,ncols)]) {
                        approximation[idx(ss*SW+i,sr*SW+j,sc*SW+k,nrows,ncols)] = x[idx(i+ds,j+dr,k+dc,nnr,nnc)];
                        float e = abs(x[idx(i+ds,j+dr,k+dc,nnr,nnc)] - data[idx(ss*SW+i,sr*SW+j,sc*SW+k,nrows,ncols)]);
                        if (e > maxe) {
                            maxe = e;
                            sw = i;
                            rw = j;
                            cw = k;
                        }
                    }
        if (maxe > target) {
            int iw = idx(ss*SW+sw,sr*SW+rw,sc*SW+cw,nrows,ncols);
            count++;
            mask[iw] = 1;
            known.push_back(make_float4(sw+ds, rw+dr, cw+dc, data[iw]));
            std::cout << ',' << ss << ',' << sr << ',' << sc << ',' << maxe << std::endl;
            for (int si = max(ss-1,0); si <= min(ss+1,sns-1); si++)
                for (int sj = max(sr-1,0); sj <= min(sr+1,snr-1); sj++)
                    for (int sk = max(sc-1,0); sk <= min(sc+1,snc-1); sk++)
                        if (!(si==ss && sj==sr && sk==sc)) {
                            bool c1 = ss*SW+sw >= max(si*SW-OW, 0) && ss*SW+sw < min((si+1)*SW+OW, nslcs);
                            bool c2 = sr*SW+rw >= max(sj*SW-OW, 0) && sr*SW+rw < min((sj+1)*SW+OW, nrows);
                            bool c3 = sc*SW+cw >= max(sk*SW-OW, 0) && sc*SW+cw < min((sk+1)*SW+OW, ncols);
                            if (c1 && c2 && c3)
                                pending[idx(si,sj,sk,snr,snc)] = true;
                        }
        } else {
            pending[idx(ss,sr,sc,snr,snc)] = false;
            break;
        }
    }
    
    return count;
}

// Insert a serial number into a filename that ends with ".txt"
inline std::string serial_name(char *name, float sn)
{
    std::string namestr = std::string(name);
    std::ostringstream oss;
    oss << sn;
    return namestr.substr(0, namestr.size()-4) + "-" + oss.str() + ".txt";
}

// Arguments: data nzmask nslcs nrows ncols mask values approximation
int main(int argc, char **argv)
{
    cusp::detail::timer timer = cusp::detail::timer();
    timer.start();
    
    const int nslcs = atoi(argv[3]);
    const int nrows = atoi(argv[4]);
    const int ncols = atoi(argv[5]);
    assert(!(nslcs%SW) && !(nrows%SW) && !(ncols%SW));
    const int snr   = nrows/SW;
    const int snc   = ncols/SW;
    const int size  = nslcs*nrows*ncols;
    const int ssize = size/SW/SW/SW;
    
    cusp::array1d<float,HOST> data(size);
    std::ifstream ifs(argv[1]);
    for (int i = 0; i < size; i++)
        ifs >> data[i];
    ifs.close();
    cusp::array1d<short,HOST> nzmask(size);
    ifs.open(argv[2]);
    for (int i = 0; i < size; i++)
        ifs >> nzmask[i];
    ifs.close();
    float vmin = 999999;
    float vmax = -999999;
    for (int i = 0; i < size; i++)
        if (nzmask[i]) {
            vmin = min(vmin, data[i]);
            vmax = max(vmax, data[i]);
        }
    float scale = 255/(vmax-vmin);
    for (int i = 0; i < size; i++)
        if (nzmask[i])
            data[i] = round((data[i]-vmin)*scale);
    std::cout << std::setprecision(7) << "Min: " << vmin << "\nMax: " << vmax << "\nScale: " << scale << std::endl;
    
    cusp::array1d<short,HOST> mask(size, 0);
    if (1) {
        for (int i = RW/2; i < nslcs; i += RW)
            for (int j = RW/2; j < nrows; j += RW)
                for (int k = RW/2; k < ncols; k += RW)
                    if (nzmask[idx(i,j,k,nrows,ncols)])
                        mask[idx(i,j,k,nrows,ncols)] = 1;
    } else {
        ifs.open("m.txt");
        for (int i = 0; i < size; i++)
            ifs >> mask[i];
        ifs.close();
    }
    cusp::array1d<bool,HOST> pending(ssize, true);
    cusp::array1d<float,HOST> x(size, 0);
    cusp::array1d<int,HOST> id(ssize);
    for (int i = 0; i < ssize; i++)
        id[i] = i;
    
    for (float percent = 3; percent >= 0.5; percent -= 0.5) {
        int count = 1;
        while (count) {
            count = 0;
            std::random_shuffle(id.begin(), id.end());
            for (int i = 0; i < ssize; i++)
                count += process_segment(percent/100*255, id[i]/(snr*snc), id[i]%(snr*snc)/snc, id[i]%snc, pending, nslcs, nrows, ncols, data, nzmask, mask, x);
            std::cout << "Added " << count << " known points" << std::endl;
        }
        
        std::ofstream ofs(serial_name(argv[6],percent));
        ofs << std::setprecision(7);
        for (int i = 0; i < size; i++)
            ofs << mask[i] << ((i+1)%ncols ? ' ' : '\n');
        ofs.close();
        ofs.open(serial_name(argv[7],percent));
        for (int i = 0; i < size; i++)
            if (mask[i])
                ofs << data[i] << '\n';
        ofs.close();
        ofs.open(serial_name(argv[8],percent));
        for (int i = 0; i < size; i++)
            ofs << min(max(x[i],0.0),255.0)/scale+vmin << ((i+1)%ncols ? ' ' : '\n');
        ofs.close();
        
        timer.soft_stop();
        std::cout << "Total seconds: " << timer.total_seconds() << std::endl;
        
        for (int i = 0; i < ssize; i++)
            pending[i] = true;
    }
    
    return 0;
}
